#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, Keno Krieger, <kriegerk@uni-bremen.de>. All rights reserved.
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Mauro Bisson <maurob@nvidia.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <map>

#include "cudamacro.h"
#include "traders.cuh"

using namespace std;

#define DIV_UP(a,b)  (((a) + ((b) - 1)) / (b))
#define MIN(a,b)	(((a) < (b)) ? (a) : (b))
#define MAX(a,b)	(((a) > (b)) ? (a) : (b))

#define THREADS 128
#define BIT_X_SPIN (4)

#define BLOCK_DIMENSION_X_DEFINE (16)
#define BLOCK_DIMENSION_Y_DEFINE (16)


map<string, string> read_config_file(string config_filename, string delimiter = "=")
{
    std::ifstream config_file;
    config_file.open(config_filename);
    map<string, string> config;

    if (!config_file.is_open()) {
        std::cout << "Could not open file '" << config_filename << "'" << std::endl;
        return config;
    } else {
        int row = 0;
        std::string line = "";
        std::string key = "";

        std::cout << "Launch configuration:" << std::endl;

        while (getline(config_file, line)) {
            if (line[0] == '#' || line == "") continue;
            int delimiter_position = line.find(delimiter);

            for (int idx = 0; idx < delimiter_position; idx++) {
                if (line[idx] != ' ') key += line[idx];
            }

            std::string value = line.substr(delimiter_position + 1, line.length() - 1);
            config[key] = value;
            std::cout << '\t' << key << ": ";
            std::cout << value << std::endl;
            row++;
            key = "";
        }
        config_file.close();
        return config;
    }
}


int main(int argc, char **argv) {

	unsigned long long *d_spins = NULL;
	unsigned long long *d_black_tiles = NULL;
	unsigned long long *d_white_tiles = NULL;

	const int SPIN_X_WORD = (8 * sizeof(*d_spins)) / BIT_X_SPIN;

	hipEvent_t start, stop;
  float elapsed_time;

	float temp  = 0.666f;

	string config_filename = (argc == 1) ? "multising.conf" : argv[1];
  map<string, string> config = read_config_file(config_filename);

  const long long grid_height = std::stoll(config["grid_height"]);
  const long long grid_width = std::stoll(config["grid_width"]);
  const long long grid_depth = std::stoll(config["grid_depth"]);
  unsigned int total_updates = std::stoul(config["total_updates"]);
  unsigned long long seed = std::stoull(config["seed"]);
  float alpha = std::stof(config["alpha"]);
  float j = std::stof(config["j"]);
  float beta = std::stof(config["beta"]);

  float reduced_alpha = -2.0f * beta * alpha;
  float reduced_j = -2.0f * beta * j;


	if (!grid_width || (grid_width % 2) || ((grid_width / 2) % (2 * SPIN_X_WORD * BLOCK_DIMENSION_X_DEFINE))) {
		fprintf(stderr, "\nPlease specify an grid_width dim multiple of %d\n\n", 2 * SPIN_X_WORD * 2 * BLOCK_DIMENSION_X_DEFINE);
		exit(EXIT_FAILURE);
	}
	if (!grid_height || (grid_height % (BLOCK_DIMENSION_Y_DEFINE))) {
		fprintf(stderr, "\nPlease specify a grid_height dim multiple of %d\n\n", BLOCK_DIMENSION_Y_DEFINE);
		exit(EXIT_FAILURE);
	}

	hipDeviceProp_t props;
	CHECK_CUDA(hipGetDeviceProperties(&props, 0));
	printf("\nUsing GPU: %s, %d SMs, %d th/SM max, CC %d.%d, ECC %s\n",
		props.name, props.multiProcessorCount,
		props.maxThreadsPerMultiProcessor,
		props.major, props.minor,
		props.ECCEnabled ? "on" : "off");

	size_t words_per_row = (grid_width / 2) / SPIN_X_WORD;
	// total lattice length
	size_t total_words = 2ull * static_cast<size_t>(grid_height) * words_per_row;

	// words_per_row / 2 because each entry in the array has two components
	dim3 blocks(DIV_UP(words_per_row / 2, BLOCK_DIMENSION_X_DEFINE), DIV_UP(grid_height, BLOCK_DIMENSION_Y_DEFINE));
	dim3 threads_per_block(BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE);

	const int redBlocks = MIN(DIV_UP(total_words, THREADS), (props.maxThreadsPerMultiProcessor / THREADS) * props.multiProcessorCount);

	unsigned long long cntPos;
	unsigned long long cntNeg;
	unsigned long long *sum_d[0];

	CHECK_CUDA(hipMalloc(&d_spins, total_words * sizeof(*d_spins)));
	CHECK_CUDA(hipMemset(d_spins, 0, total_words * sizeof(*d_spins)));

	CHECK_CUDA(hipMalloc(&sum_d[0], 2 * sizeof(**sum_d)));

	d_black_tiles = d_spins;
	d_white_tiles = d_spins + total_words / 2;

	float *exp_d[0];
	float  exp_h[2][5];

	// precompute possible exponentials
	for(int i = 0; i < 2; i++) {
		for(int j = 0; j < 5; j++) {
			if(temp > 0) {
				exp_h[i][j] = expf((i ? -2.0f : 2.0f) * static_cast<float>(j * 2 - 4) * (1.0f / temp));
			} else {
				if(j == 2) {
					exp_h[i][j] = 0.5f;
				} else {
					exp_h[i][j] = (i ? -2.0f : 2.0f) * static_cast<float>(j * 2 - 4);
				}
			}
		}
	}

	CHECK_CUDA(hipSetDevice(0));
	CHECK_CUDA(hipMalloc(exp_d, 2 * 5 * sizeof(**exp_d)));
	CHECK_CUDA(hipMemcpy(exp_d[0], exp_h, 2 * 5 * sizeof(**exp_d), hipMemcpyHostToDevice));

	CHECK_CUDA(hipEventCreate(&start));
	CHECK_CUDA(hipEventCreate(&stop));

	CHECK_CUDA(hipSetDevice(0));
	initialise_traders<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BIT_X_SPIN, C_BLACK, unsigned long long>
	<<<blocks, threads_per_block>>>
	(seed, words_per_row / 2, reinterpret_cast<ulonglong2 *>(d_black_tiles));
	CHECK_ERROR("initialise_traders");

	initialise_traders<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BIT_X_SPIN, C_WHITE, unsigned long long>
	<<<blocks, threads_per_block>>>
	(seed, words_per_row / 2, reinterpret_cast<ulonglong2 *>(d_white_tiles));
	CHECK_ERROR("initialise_traders");

	// compute sum over array
	countSpins(redBlocks, total_words, d_black_tiles, d_white_tiles, sum_d, &cntPos, &cntNeg);
	printf("\nInitial magnetization: %9.6lf, up_s: %12llu, dw_s: %12llu\n",
	       abs(static_cast<double>(cntPos)-static_cast<double>(cntNeg)) / (total_words*SPIN_X_WORD),
	       cntPos, cntNeg);

	CHECK_CUDA(hipSetDevice(0));
	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipEventRecord(start, 0));
  int iteration;
	// main update loop
	for(iteration = 0; iteration < total_updates; iteration++) {

		CHECK_CUDA(hipSetDevice(0));
		update_strategies<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BIT_X_SPIN, C_BLACK, unsigned long long>
		<<<blocks, threads_per_block>>>
		(seed, iteration + 1, (grid_width / 2) / SPIN_X_WORD / 2, grid_height, words_per_row / 2,
		 reinterpret_cast<float (*)[5]>(exp_d[0]),
		 reinterpret_cast<ulonglong2 *>(d_white_tiles),
		 reinterpret_cast<ulonglong2 *>(d_black_tiles));

		CHECK_CUDA(hipSetDevice(0));
		update_strategies<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BIT_X_SPIN, C_WHITE, unsigned long long>
		<<<blocks, threads_per_block>>>
		(seed, iteration + 1, (grid_width / 2) / SPIN_X_WORD / 2, grid_height, words_per_row / 2,
		 reinterpret_cast<float (*)[5]>(exp_d[0]),
		 reinterpret_cast<ulonglong2 *>(d_black_tiles),
		 reinterpret_cast<ulonglong2 *>(d_white_tiles));
	}
	CHECK_CUDA(hipEventRecord(stop, 0));
	CHECK_CUDA(hipEventSynchronize(stop));

	// compute total sum
	countSpins(redBlocks, total_words, d_black_tiles, d_white_tiles, sum_d, &cntPos, &cntNeg);
	printf("Final   magnetization: %9.6lf, up_s: %12llu, dw_s: %12llu \n\n",
	       abs(static_cast<double>(cntPos)-static_cast<double>(cntNeg)) / (total_words*SPIN_X_WORD),
	       cntPos, cntNeg);

	CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

	printf("Kernel execution time: %E ms, %.2lf flips/ns \n",
		elapsed_time, static_cast<double>(total_words * SPIN_X_WORD) * iteration / (elapsed_time * 1.0E+6));

	CHECK_CUDA(hipFree(d_spins));
	CHECK_CUDA(hipFree(exp_d[0]));
	CHECK_CUDA(hipFree(sum_d[0]));

  CHECK_CUDA(hipSetDevice(0));
  CHECK_CUDA(hipDeviceReset());

	return 0;
}
