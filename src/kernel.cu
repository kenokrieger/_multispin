#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <map>

#include "cudamacro.h"
#include "traders.cuh"

using namespace std;

#define DIV_UP(a,b)  (((a) + ((b) - 1)) / (b))
#define MIN(a,b)	(((a) < (b)) ? (a) : (b))
#define MAX(a,b)	(((a) > (b)) ? (a) : (b))

#define THREADS 128
#define BIT_X_SPIN (4)

#define THREADS_X (16)
#define THREADS_Y (16)


#define MAG_FILE_NAME "magnetisation.dat"
#define LOG_FILE_NAME "log"


map<string, string> read_config_file(string config_filename, string delimiter = "=")
{
    std::ifstream config_file;
    config_file.open(config_filename);
    map<string, string> config;

    if (!config_file.is_open()) {
        std::cout << "Could not open config file '" << config_filename << "'" << std::endl;
		    exit(EXIT_FAILURE);
    } else {
        int row = 0;
        std::string line = "";
        std::string key = "";

        std::cout << "Launch configuration:" << std::endl;

        while (getline(config_file, line)) {
            if (line[0] == '#' || line == "") continue;
            int delimiter_position = line.find(delimiter);

            for (int idx = 0; idx < delimiter_position; idx++) {
                if (line[idx] != ' ') key += line[idx];
            }

            std::string value = line.substr(delimiter_position + 1, line.length() - 1);
            config[key] = value;
            std::cout << '\t' << key << ": ";
            std::cout << value << std::endl;
            row++;
            key = "";
        }
        config_file.close();
        return config;
    }
}


void validate_grid(const long long grid_width, const long long grid_height,
                   const int spin_x_word)
{
	if (!grid_width || (grid_width % 2) || ((grid_width / 2) % (2 * spin_x_word * THREADS_X))) {
		fprintf(stderr, "\nPlease specify an grid_width multiple of %d\n\n", 2 * spin_x_word * 2 * THREADS_X);
		exit(EXIT_FAILURE);
	}
	if (!grid_height || (grid_height % (THREADS_Y))) {
		fprintf(stderr, "\nPlease specify a grid_height multiple of %d\n\n", THREADS_Y);
		exit(EXIT_FAILURE);
	}
}


hipDeviceProp_t identify_gpu()
{
  hipDeviceProp_t props;
  CHECK_CUDA(hipGetDeviceProperties(&props, 0));
  printf("\nUsing GPU: %s, %d SMs, %d th/SM max, CC %d.%d, ECC %s\n",
    props.name, props.multiProcessorCount,
    props.maxThreadsPerMultiProcessor,
    props.major, props.minor,
    props.ECCEnabled ? "on" : "off");
  return props;
}


char* getDefaultConfigName(char* path)
{
  int last_slash = 0;
  for (int idx = strlen(path) - 1; idx > 0; idx--) {
    if (path[idx] == '/') {
      last_slash = idx;
      break;
    }
  }
  char *config_name = (char *) malloc(strlen(path) + 5 * sizeof(char));
  strcat(config_name, path + last_slash + 1);
  strcat(config_name, ".conf");
  return config_name;
}


int main(int argc, char **argv) {

	unsigned long long *d_spins = NULL;
	unsigned long long *d_black_tiles = NULL;
	unsigned long long *d_white_tiles = NULL;

	const int SPIN_X_WORD = (8 * sizeof(*d_spins)) / BIT_X_SPIN;

	hipEvent_t start, stop;
  float elapsed_time;

	string config_filename = (argc == 1) ? getDefaultConfigName(argv[0]) : argv[1];
  map<string, string> config = read_config_file(config_filename);

  const long long grid_height = std::stoll(config["grid_height"]);
  const long long grid_width = std::stoll(config["grid_width"]);
  const unsigned int total_updates = std::stoul(config["total_updates"]);
  const unsigned long long seed = std::stoull(config["seed"]);
  float alpha = std::stof(config["alpha"]);
  float j = std::stof(config["j"]);
  float beta = std::stof(config["beta"]);
  float percentage_up = std::stof(config["init_up"]);
  int device_id = std::stoi(config["GPU"]);

  const float reduced_alpha = -2.0f * beta * alpha;
  const float reduced_j = -2.0f * beta * j;

	validate_grid(grid_width, grid_height, SPIN_X_WORD);

  hipDeviceProp_t props = identify_gpu();

	const size_t words_per_row = (grid_width / 2) / SPIN_X_WORD;
	const size_t total_words = 2ull * static_cast<size_t>(grid_height) * words_per_row;

  // words_per_row / 2 because each entry in the array has two components
  dim3 blocks(DIV_UP(words_per_row / 2, THREADS_X), DIV_UP(grid_height, THREADS_Y));
  dim3 threads_per_block(THREADS_X, THREADS_Y);
	const int reduce_blocks = MIN(DIV_UP(total_words, THREADS), (props.maxThreadsPerMultiProcessor / THREADS) * props.multiProcessorCount);

	unsigned long long spins_up;
	unsigned long long spins_down;
	unsigned long long *d_sum;

	CHECK_CUDA(hipMalloc(&d_spins, total_words * sizeof(*d_spins)));
	CHECK_CUDA(hipMemset(d_spins, 0, total_words * sizeof(*d_spins)));

	CHECK_CUDA(hipMalloc(&d_sum, 2 * sizeof(*d_sum)));

	d_black_tiles = d_spins;
	d_white_tiles = d_spins + total_words / 2;

	float *d_probabilities;
	CHECK_CUDA(hipMalloc(&d_probabilities, 2 * 5 * sizeof(*d_probabilities)));

	CHECK_CUDA(hipEventCreate(&start));
	CHECK_CUDA(hipEventCreate(&stop));

  // const long long total_threads = blocks.x * blocks.y * blocks.z * threads_per_block.x * threads_per_block.y * threads_per_block.z;
  // hiprandStatePhilox4_32_10_t* states;
  // hipMalloc((void**) &states, total_threads * sizeof(hiprandStatePhilox4_32_10_t));

	initialise_arrays<unsigned long long>(device_id, blocks, threads_per_block,
                                        seed, words_per_row / 2,
                                        d_black_tiles, d_white_tiles, percentage_up);

	CHECK_CUDA(hipSetDevice(0));
	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipEventRecord(start, 0));
  int iteration;
  float global_market = 0;
	std::ofstream magfile;

  magfile.open(MAG_FILE_NAME);
	for(iteration = 0; iteration < total_updates; iteration++) {
		global_market = update<SPIN_X_WORD>(
      device_id, 
      iteration, blocks, threads_per_block, reduce_blocks,
			d_black_tiles, d_white_tiles, d_sum, d_probabilities,
			spins_up, spins_down,
			seed, reduced_alpha, reduced_j,
	    grid_height, grid_width,
			words_per_row, total_words
    );
    std::cout << "\r" << global_market << std::flush;
  	magfile << global_market << std::endl;
	}
  std::cout << std::endl;
  magfile.close();

	CHECK_CUDA(hipEventRecord(stop, 0));
	CHECK_CUDA(hipEventSynchronize(stop));

	CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

  double spin_updates_per_nanosecond = static_cast<double>(total_words * SPIN_X_WORD) * iteration / (elapsed_time * 1.0E+6);
	printf("Kernel execution time: %.2f s, %.2lf flips/ns \n", elapsed_time * 1.0E-3, spin_updates_per_nanosecond);
  printf("Final magnetisation: %f\n", global_market);

  std::ofstream logfile;
  logfile.open(LOG_FILE_NAME);
  logfile << "updates/ns: " << spin_updates_per_nanosecond << std::endl;
  logfile.close();

	CHECK_CUDA(hipFree(d_spins));
	CHECK_CUDA(hipFree(d_probabilities));
	CHECK_CUDA(hipFree(d_sum));

  CHECK_CUDA(hipSetDevice(0));
  CHECK_CUDA(hipDeviceReset());

	return 0;
}
