#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, Keno Krieger, <kriegerk@uni-bremen.de>. All rights reserved.
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Mauro Bisson <maurob@nvidia.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "cudamacro.h"

#define DIV_UP(a,b)  (((a) + ((b) - 1)) / (b))
#define MIN(a,b)	(((a) < (b)) ? (a) : (b))
#define MAX(a,b)	(((a) > (b)) ? (a) : (b))

#define THREADS 128
#define BIT_X_SPIN (4)

/*
 * 2048+: 16, 16, 2, 1
 *  1024: 16, 16, 1, 2
 *   512:  8,  8, 1, 1
 *   256:  4,  8, 1, 1
 *   128:  2,  8, 1, 1
*/
#define BLOCK_DIMENSION_X_DEFINE (16)
#define BLOCK_DIMENSION_Y_DEFINE (16)

#define TOTAL_UPDATES_DEFAULT (10000)
#define SEED_DEFAULT  (463463564571ull)


__device__ __forceinline__ unsigned int __custom_popc(const unsigned int x) {return __popc(x);}

__device__ __forceinline__ unsigned long long int __custom_popc(const unsigned long long int x) {return __popcll(x);}

enum {C_BLACK, C_WHITE};

// creates a vector with two components
__device__ __forceinline__ uint2 __custom_make_int2(const unsigned int x, const unsigned int y) {return make_uint2(x, y);}

__device__ __forceinline__ ulonglong2 __custom_make_int2(const unsigned long long x, const unsigned long long y) {return make_ulonglong2(x, y);}


template<int BLOCK_DIMENSION_X, int BLOCK_DIMENSION_Y, int BITXSPIN, int COLOR, typename INT_T, typename INT2_T>
__global__  void initialise_traders(const long long seed, const long long number_of_columns, INT2_T *__restrict__ traders)
{
	const int row = blockIdx.y * BLOCK_DIMENSION_Y + threadIdx.y;
	const int col = blockIdx.x * BLOCK_DIMENSION_X + threadIdx.x;
  const int index = row * number_of_columns + col;
	const int SPIN_X_WORD = 8 * sizeof(INT_T) / BITXSPIN;

	const long long thread_id = ((gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x) * BLOCK_DIMENSION_X * BLOCK_DIMENSION_Y +
	                              threadIdx.y * BLOCK_DIMENSION_X + threadIdx.x;

	hiprandStatePhilox4_32_10_t rng;
	hiprand_init(seed, thread_id, static_cast<long long>(2 * SPIN_X_WORD) * COLOR, &rng);

  traders[index] = __custom_make_int2(INT_T(0), INT_T(0));
	for(int spin_position = 0; spin_position < 8 * sizeof(INT_T); spin_position += BITXSPIN) {
		// The two if clauses are not identical since hiprand_uniform()
		// returns a different number on each invokation
    /*
     * shift the spin with value 1 to its respective position and then
     * assign the matching bit the value 1 by using the bitwise
     * logical or operator |=
     * shift: 0000000000000000001 -> 0000000000010000000
     * logical bitwise or with traders:
     * traders =                  0000000000000001000
     * INT_T(1) << spin_position = 0000000000010000000
     * =>  traders =              0000000000010001000
     */
		if (hiprand_uniform(&rng) < 0.5f) {
			traders[index].x |= INT_T(1) << spin_position;
		}
		if (hiprand_uniform(&rng) < 0.5f) {
			traders[index].y |= INT_T(1) << spin_position;
		}
	}
	return;
}


template<int TILE_SIZE_X, int TILE_SIZE_Y, typename INT2_T>
__device__ void load_tiles(const int grid_width, const int grid_height, const long long number_of_columns,
                           const INT2_T *__restrict__ traders, INT2_T tile[][TILE_SIZE_X + 2])
    /*
    Each threads_per_block works on one tile with shape (TILE_SIZE_Y + 2, TILE_SIZE_X + 2).
    */
{
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	const int tile_start_x = blockIdx.x * TILE_SIZE_X;
	const int tile_start_y = blockIdx.y * TILE_SIZE_Y;

	int row = tile_start_y + tidy;
	int col = tile_start_x + tidx;
	tile[1 + tidy][1 + tidx] = traders[row * number_of_columns + col];

	if (tidy == 0) {
		row = (tile_start_y % grid_height) == 0 ? tile_start_y + grid_height - 1 : tile_start_y - 1;
		tile[0][1 + tidx] = traders[row * number_of_columns + col];

		row = ((tile_start_y + TILE_SIZE_Y) % grid_height) == 0 ? tile_start_y + TILE_SIZE_Y - grid_height : tile_start_y + TILE_SIZE_Y;
		tile[1 + TILE_SIZE_Y][1 + tidx] = traders[row * number_of_columns + col];

		row = tile_start_y + tidx;
		col = (tile_start_x % grid_width) == 0 ? tile_start_x + grid_width - 1 : tile_start_x - 1;
		tile[1 + tidx][0] = traders[row * number_of_columns + col];

		row = tile_start_y + tidx;
		col = ((tile_start_x + TILE_SIZE_X) % grid_width) == 0 ? tile_start_x + TILE_SIZE_X - grid_width : tile_start_x + TILE_SIZE_X;
		tile[1 + tidx][1 + TILE_SIZE_X] = traders[row * number_of_columns + col];
	}
	return;
}


__device__ void load_probabilities(const float precomputed_probabilities[][5], float shared_probabilities[2][5],
                                   const int block_dimension_x, const int block_dimension_y,
                                   const int tidx, const int tidy)
{
  // load precomputed exponentials into shared memory.
  // in case a threads_per_block consists of less than 2 * 5 threads
  // multiple iterations in each thread are needed
  // otherwise loops will only trigger once
  #pragma unroll
  for(int i = 0; i < 2; i += block_dimension_x) {
    #pragma unroll
    for(int j = 0; j < 5; j += block_dimension_y) {
      if (i + tidy < 2 && j + tidx < 5) {
        shared_probabilities[i + tidy][j + tidx] = precomputed_probabilities[i + tidy][j + tidx];
      }
    }
  }
  return;
}


template<int BLOCK_DIMENSION_X, int BLOCK_DIMENSION_Y, int BITXSPIN, int COLOR, typename INT_T, typename INT2_T>
__global__ void update_strategies(const long long seed, const int number_of_previous_iterations,
		       const int grid_width, // lattice width of one color in words
		       const int grid_height, // lattice height (not in words)
		       const long long number_of_columns,
		       const float precomputed_probabilities[][5],
		       const INT2_T *__restrict__ checkerboard_agents,
		             INT2_T *__restrict__ traders)
{
	const int SPIN_X_WORD = 8 * sizeof(INT_T) / BITXSPIN;
	const INT_T ONE = static_cast<INT_T>(1);
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	__shared__ INT2_T shared_tiles[BLOCK_DIMENSION_Y + 2][BLOCK_DIMENSION_X + 2];
	load_tiles<BLOCK_DIMENSION_X, BLOCK_DIMENSION_Y, INT2_T>
  (grid_width, grid_height, number_of_columns, checkerboard_agents, shared_tiles);

	__shared__ float shared_probabilities[2][5];
  load_probabilities(precomputed_probabilities, shared_probabilities, BLOCK_DIMENSION_X, BLOCK_DIMENSION_Y, tidx, tidy);

	__syncthreads();


	const int row = blockIdx.y * BLOCK_DIMENSION_Y + tidy;
	const int col = blockIdx.x * BLOCK_DIMENSION_X + tidx;

	const long long thread_id = (blockIdx.y * gridDim.x + blockIdx.x) * BLOCK_DIMENSION_X * BLOCK_DIMENSION_Y
                            +  threadIdx.y * BLOCK_DIMENSION_X + threadIdx.x;

	INT2_T target = traders[row * number_of_columns + col];

	// three nearest neighbors
	INT2_T upper_neighbor = shared_tiles[    tidy][1 + tidx];
	INT2_T center_neighbor = shared_tiles[1 + tidy][1 + tidx];
	INT2_T lower_neighbor = shared_tiles[2 + tidy][1 + tidx];

	const int shift_left = (COLOR == C_BLACK) ? !(row % 2) : (row % 2);
	// remaining neighbor, either left or right
	INT2_T horizontal_neighbor = (shift_left) ? shared_tiles[1 + tidy][tidx] : shared_tiles[1 + tidy][2 + tidx];

	if (shift_left) {
  	horizontal_neighbor.x = (center_neighbor.x << BITXSPIN) | (horizontal_neighbor.y >> (8 * sizeof(horizontal_neighbor.y) - BITXSPIN));
  	horizontal_neighbor.y = (center_neighbor.y << BITXSPIN) | (center_neighbor.x >> (8 * sizeof(center_neighbor.x) - BITXSPIN));
	} else {
		horizontal_neighbor.y = (center_neighbor.y >> BITXSPIN) | (horizontal_neighbor.x << (8 * sizeof(horizontal_neighbor.x) - BITXSPIN));
		horizontal_neighbor.x = (center_neighbor.x >> BITXSPIN) | (center_neighbor.y << (8 * sizeof(center_neighbor.y) - BITXSPIN));
	}

	hiprandStatePhilox4_32_10_t rng;
	hiprand_init(seed, thread_id, static_cast<long long>(2 * SPIN_X_WORD) * (2 * number_of_previous_iterations + COLOR), &rng);

	// this basically sums over all spins/word in parallel
	center_neighbor.x += upper_neighbor.x + lower_neighbor.x + horizontal_neighbor.x;
	center_neighbor.y += upper_neighbor + lower_neighbor.y + horizontal_neighbor.y;

	for(int spin_position = 0; spin_position < 8 * sizeof(INT_T); spin_position += BITXSPIN) {

		// convert binary values to hexadecimal values, basically mapping 0, 2, 4, 6, 8
		// onto 0, 1, 2, 3, 4 for easier array access
		const int2 source = make_int2((target.x >> spin_position) & 0xF, (target.y >> spin_position) & 0xF);
		const int2 sum = make_int2((center_neighbor.x >> spin_position) & 0xF, (center_neighbor.y >> spin_position) & 0xF);

		if (hiprand_uniform(&rng) <= shared_probabilities[source.x][sum.x]) {
			target.x |= ONE << spin_position;
		}
		if (hiprand_uniform(&rng) <= shared_probabilities[source.y][sum.y]) {
			target.y |= ONE << spin_position;
		}
	}

	traders[row * number_of_columns + col] = target;

	return;
}


template<int BLOCK_DIMENSION_X, int WSIZE, typename T>
__device__ __forceinline__ T __block_sum(T traders)
{
	__shared__ T sh[BLOCK_DIMENSION_X / WSIZE];

	const int lid = threadIdx.x%WSIZE;
	const int wid = threadIdx.x/WSIZE;

	#pragma unroll
	for(int i = WSIZE/2; i; i >>= 1) {
		traders += __shfl_down_sync(0xFFFFFFFF, traders, i);
	}
	if (lid == 0) sh[wid] = traders;

	__syncthreads();
	if (wid == 0) {
		traders = (lid < (BLOCK_DIMENSION_X / WSIZE)) ? sh[lid] : 0;

		#pragma unroll
		for(int i = (BLOCK_DIMENSION_X/WSIZE)/2; i; i >>= 1) {
			traders += __shfl_down_sync(0xFFFFFFFF, traders, i);
		}
	}
	__syncthreads();
	return traders;
}

// to be optimized
template<int BLOCK_DIMENSION_X, int BITXSPIN, typename INT_T, typename SUM_T>
__global__ void getMagn_k(const long long n,
			                    const INT_T *__restrict__ traders,
			                    SUM_T *__restrict__ sum)
{

	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSPIN;

	const long long nth = static_cast<long long>(blockDim.x)*gridDim.x;
	const long long thread_id = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

	SUM_T __cntP = 0;
	SUM_T __cntN = 0;

	for(long long i = 0; i < n; i += nth) {
		if (i+thread_id < n) {
			const int __c = __custom_popc(traders[i+thread_id]);
			__cntP += __c;
			__cntN += SPIN_X_WORD - __c;
		}
	}
	__cntP = __block_sum<BLOCK_DIMENSION_X, 32>(__cntP);
	__cntN = __block_sum<BLOCK_DIMENSION_X, 32>(__cntN);

	if (threadIdx.x == 0) {
		atomicAdd(sum + 0, __cntP);
		atomicAdd(sum + 1, __cntN);
	}
	return;
}


static void countSpins(const int redBlocks,
								       const size_t total_words,
								       const unsigned long long *d_black_tiles,
								       const unsigned long long *d_white_tiles,
									     unsigned long long **sum_d,
									     unsigned long long *bsum,
									     unsigned long long *wsum)
{
	CHECK_CUDA(hipMemset(sum_d[0], 0, 2*sizeof(**sum_d)));
	getMagn_k<THREADS, BIT_X_SPIN><<<redBlocks, THREADS>>>(total_words, d_black_tiles, sum_d[0]);
	CHECK_ERROR("getMagn_k");
	CHECK_CUDA(hipDeviceSynchronize());

	bsum[0] = 0;
	wsum[0] = 0;

	unsigned long long sum_h[0][2];

	CHECK_CUDA(hipMemcpy(sum_h[0], sum_d[0], 2*sizeof(**sum_h), hipMemcpyDeviceToHost));
	bsum[0] += sum_h[0][0];
	wsum[0] += sum_h[0][1];

	return;
}


int main(int argc, char **argv) {

	unsigned long long *d_spins = NULL;
	unsigned long long *d_black_tiles = NULL;
	unsigned long long *d_white_tiles = NULL;

	hipEvent_t start, stop;
  float elapsed_time;

	const int SPIN_X_WORD = (8 * sizeof(*d_spins)) / BIT_X_SPIN;
	int grid_width = 2048;
	int grid_height = 2048;

	int total_updates = TOTAL_UPDATES_DEFAULT;

	unsigned long long seed = SEED_DEFAULT;

	float temp  = 0.666f;

	if (!grid_width || (grid_width % 2) || ((grid_width / 2) % (2 * SPIN_X_WORD * BLOCK_DIMENSION_X_DEFINE))) {
		fprintf(stderr, "\nPlease specify an grid_width dim multiple of %d\n\n", 2 * SPIN_X_WORD * 2 * BLOCK_DIMENSION_X_DEFINE);
		exit(EXIT_FAILURE);
	}
	if (!grid_height || (grid_height % (BLOCK_DIMENSION_Y_DEFINE))) {
		fprintf(stderr, "\nPlease specify a grid_height dim multiple of %d\n\n", BLOCK_DIMENSION_Y_DEFINE);
		exit(EXIT_FAILURE);
	}

	hipDeviceProp_t props;

	printf("\nUsing GPU:\n");

	CHECK_CUDA(hipGetDeviceProperties(&props, 0));
	printf("\t%2d (%s, %d SMs, %d th/SM max, CC %d.%d, ECC %s)\n",
		0, props.name, props.multiProcessorCount,
		props.maxThreadsPerMultiProcessor,
		props.major, props.minor,
		props.ECCEnabled ? "on" : "off");

	printf("\n");

	size_t words_per_row = (grid_width / 2) / SPIN_X_WORD;
	// total lattice length
	size_t total_words = 2ull * static_cast<size_t>(grid_height) * words_per_row;

	// words_per_row / 2 because each entry in the array has two components
	dim3 blocks(DIV_UP(words_per_row / 2, BLOCK_DIMENSION_X_DEFINE), DIV_UP(grid_height, BLOCK_DIMENSION_Y_DEFINE));
	dim3 threads_per_block(BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE);

	printf("Run configuration:\n");
	printf("\tspin/word: %d\n", SPIN_X_WORD);
	printf("\tspins: %zu\n", total_words * SPIN_X_WORD);
	printf("\tseed: %llu\n", seed);
	printf("\titerations: %d\n", total_updates);
	printf("\tblock (x, y): %d, %d\n", threads_per_block.x, threads_per_block.y);
	printf("\ttile  (x, y): %d, %d\n", BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE);
	printf("\tgrid  (x, y): %d, %d\n", blocks.x, blocks.y);

	printf("\ttemp: %f \n", temp);

	printf("\n");

	printf("\tlattice size:      %8d x %8d\n", grid_height, grid_width);
	printf("\tlattice shape: 2 x %8d x %8zu (%12zu %s)\n", grid_height, words_per_row, total_words, sizeof(*d_spins) == 4 ? "uints" : "ulls");
	printf("\tmemory: %.2lf MB \n", (total_words * sizeof(*d_spins)) / (1024.0 * 1024.0));

	const int redBlocks = MIN(DIV_UP(total_words, THREADS), (props.maxThreadsPerMultiProcessor / THREADS) * props.multiProcessorCount);

	unsigned long long cntPos;
	unsigned long long cntNeg;
	unsigned long long *sum_d[0];

	CHECK_CUDA(hipMalloc(&d_spins, total_words * sizeof(*d_spins)));
	CHECK_CUDA(hipMemset(d_spins, 0, total_words * sizeof(*d_spins)));

	CHECK_CUDA(hipMalloc(&sum_d[0], 2 * sizeof(**sum_d)));


	d_black_tiles = d_spins;
	d_white_tiles = d_spins + total_words / 2;

	float *exp_d[0];
	float  exp_h[2][5];

	// precompute possible exponentials
	for(int i = 0; i < 2; i++) {
		for(int j = 0; j < 5; j++) {
			if(temp > 0) {
				exp_h[i][j] = expf((i ? -2.0f : 2.0f) * static_cast<float>(j * 2 - 4) * (1.0f / temp));
			} else {
				if(j == 2) {
					exp_h[i][j] = 0.5f;
				} else {
					exp_h[i][j] = (i ? -2.0f : 2.0f) * static_cast<float>(j * 2 - 4);
				}
			}
		}
	}

	CHECK_CUDA(hipSetDevice(0));
	CHECK_CUDA(hipMalloc(exp_d, 2 * 5 * sizeof(**exp_d)));
	CHECK_CUDA(hipMemcpy(exp_d[0], exp_h, 2 * 5 * sizeof(**exp_d), hipMemcpyHostToDevice));

	CHECK_CUDA(hipEventCreate(&start));
	CHECK_CUDA(hipEventCreate(&stop));

	CHECK_CUDA(hipSetDevice(0));
	initialise_traders<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BIT_X_SPIN, C_BLACK, unsigned long long>
	<<<blocks, threads_per_block>>>
	(seed, words_per_row / 2, reinterpret_cast<ulonglong2 *>(d_black_tiles));
	CHECK_ERROR("initialise_traders");

	initialise_traders<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BIT_X_SPIN, C_WHITE, unsigned long long>
	<<<blocks, threads_per_block>>>
	(seed, words_per_row / 2, reinterpret_cast<ulonglong2 *>(d_white_tiles));
	CHECK_ERROR("initialise_traders");

	// compute sum over array
	countSpins(redBlocks, total_words, d_black_tiles, d_white_tiles, sum_d, &cntPos, &cntNeg);
	printf("\nInitial magnetization: %9.6lf, up_s: %12llu, dw_s: %12llu\n",
	       abs(static_cast<double>(cntPos)-static_cast<double>(cntNeg)) / (total_words*SPIN_X_WORD),
	       cntPos, cntNeg);

	CHECK_CUDA(hipSetDevice(0));
	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipEventRecord(start, 0));
  int iteration;
	// main update loop
	for(iteration = 0; iteration < total_updates; iteration++) {

		CHECK_CUDA(hipSetDevice(0));
		update_strategies<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BIT_X_SPIN, C_BLACK, unsigned long long>
		<<<blocks, threads_per_block>>>
		(seed, iteration + 1, (grid_width / 2) / SPIN_X_WORD / 2, grid_height, words_per_row / 2,
		 reinterpret_cast<float (*)[5]>(exp_d[0]),
		 reinterpret_cast<ulonglong2 *>(d_white_tiles),
		 reinterpret_cast<ulonglong2 *>(d_black_tiles));

		CHECK_CUDA(hipSetDevice(0));
		update_strategies<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BIT_X_SPIN, C_WHITE, unsigned long long>
		<<<blocks, threads_per_block>>>
		(seed, iteration + 1, (grid_width / 2) / SPIN_X_WORD / 2, grid_height, words_per_row / 2,
		 reinterpret_cast<float (*)[5]>(exp_d[0]),
		 reinterpret_cast<ulonglong2 *>(d_black_tiles),
		 reinterpret_cast<ulonglong2 *>(d_white_tiles));
	}
	CHECK_CUDA(hipEventRecord(stop, 0));
	CHECK_CUDA(hipEventSynchronize(stop));

	// compute total sum
	countSpins(redBlocks, total_words, d_black_tiles, d_white_tiles, sum_d, &cntPos, &cntNeg);
	printf("Final   magnetization: %9.6lf, up_s: %12llu, dw_s: %12llu (iter: %8d)\n\n",
	       abs(static_cast<double>(cntPos)-static_cast<double>(cntNeg)) / (total_words*SPIN_X_WORD),
	       cntPos, cntNeg, iteration);

	CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

	printf("Kernel execution time for %d update steps: %E ms, %.2lf flips/ns (BW: %.2lf GB/s)\n",
		iteration, elapsed_time, static_cast<double>(total_words * SPIN_X_WORD) * iteration / (elapsed_time * 1.0E+6),
		(2ull * iteration * (
			  sizeof(*d_spins)*((total_words / 2) + (total_words / 2) + (total_words / 2))  // src color read, dst color read, dst color write
			+ sizeof(*exp_d) * 5 * blocks.x * blocks.y ) * 1.0E-9) / (elapsed_time / 1.0E+3));

	CHECK_CUDA(hipFree(d_spins));
	CHECK_CUDA(hipFree(exp_d[0]));
	CHECK_CUDA(hipFree(sum_d[0]));

  CHECK_CUDA(hipSetDevice(0));
  CHECK_CUDA(hipDeviceReset());

	return 0;
}
