#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Mauro Bisson <maurob@nvidia.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <getopt.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "cudamacro.h" /* for time() */
#include "utils.h"

#define DIV_UP(a,b)     (((a)+((b)-1))/(b))

#define THREADS  128

#define BIT_X_SPIN (4)

#define CRIT_TEMP	(2.26918531421f)
#define	ALPHA_DEF	(0.1f)
#define MIN_TEMP	(0.05f*CRIT_TEMP)

#define MIN(a,b)	(((a)<(b))?(a):(b))
#define MAX(a,b)	(((a)>(b))?(a):(b))

// 2048+: 16, 16, 2, 1
//  1024: 16, 16, 1, 2
//   512:  8,  8, 1, 1
//   256:  4,  8, 1, 1
//   128:  2,  8, 1, 1

#define BLOCK_DIMENSION_X_DEFINE (16)
#define BLOCK_DIMENSION_Y_DEFINE (16)
#define BMULT_X (2)
#define BMULT_Y (1)

#define MAX_GPU	(256)

#define TOTAL_UPDATES_DEFAULT (10000)
#define SEED_DEFAULT  (463463564571ull)

#define MAX_CORR_LEN (128)

__device__ __forceinline__ unsigned int __mypopc(const unsigned int x) {return __popc(x);}

__device__ __forceinline__ unsigned long long int __mypopc(const unsigned long long int x) {return __popcll(x);}

enum {C_BLACK, C_WHITE};

// creates a vector with two components
__device__ __forceinline__ uint2 __mymake_int2(const unsigned int x, const unsigned int y) {return make_uint2(x, y);}

__device__ __forceinline__ ulonglong2 __mymake_int2(const unsigned long long x, const unsigned long long y) {return make_ulonglong2(x, y);}


template<int BLOCK_DIMENSION_X, int BLOCK_DIMENSION_Y, int LOOP_X, int LOOP_Y, int BITXSPIN, int COLOR, typename INT_T, typename INT2_T>
__global__  void initialise_traders(const long long seed, const long long dimX, INT2_T *__restrict__ traders)
{
	const int row = blockIdx.y * BLOCK_DIMENSION_Y * LOOP_Y + threadIdx.y;
	const int col = blockIdx.x * BLOCK_DIMENSION_X * LOOP_X + threadIdx.x;

	const int SPIN_X_WORD = 8 * sizeof(INT_T) / BITXSPIN;

	const long long thread_id = ((gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x) * BLOCK_DIMENSION_X * BLOCK_DIMENSION_Y +
	                              threadIdx.y * BLOCK_DIMENSION_X + threadIdx.x;

	hiprandStatePhilox4_32_10_t rng;
	hiprand_init(seed, thread_id, static_cast<long long>(2 * SPIN_X_WORD) * LOOP_X * LOOP_Y * COLOR, &rng);

  // fill temporary 2d-array with 2d-vectors where both components are 0
	INT2_T __tmp[LOOP_Y][LOOP_X];
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__tmp[i][j] = __mymake_int2(INT_T(0),INT_T(0));
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			#pragma unroll
			for(int bit_position = 0; bit_position < 8 * sizeof(INT_T); bit_position += BITXSPIN) {
				// These two if clauses are not identical since hiprand_uniform()
				// returns a different number on each invokation
				if (hiprand_uniform(&rng) < 0.5f) {
          /*
           * shift the spin 1 to its respective position and then assign the
           * matching bit the value 1 by using the bitwise logical or |=
           * shift: 0000000000000000001 -> 0000000000010000000
           * logical bitwise or with tmp:
           * tmp[i][j] =                0000000000000001000
           * INT_T(1) << bit_position = 0000000000010000000
           * =>  tmp[i][j] =            0000000000010001000
           */
					__tmp[i][j].x |= INT_T(1) << bit_position;
				}
				if (hiprand_uniform(&rng) < 0.5f) {
					__tmp[i][j].y |= INT_T(1) << bit_position;
				}
			}
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			traders[(row + i * BLOCK_DIMENSION_Y) * dimX + col + j * BLOCK_DIMENSION_X] = __tmp[i][j];
		}
	}
	return;
}

template<int BLOCK_DIMENSION_X,
	 int BLOCK_DIMENSION_Y,
	 int LOOP_X,
	 int LOOP_Y,
	 int BITXSPIN,
	 typename INT_T,
	 typename INT2_T>
__global__  void hamiltInitB_k(const int device_id,
			       const float tgtProb,
			       const long long seed,
                               const long long begY,
                               const long long dimX, // ld
                                     INT2_T *__restrict__ hamB) {

	const int row = blockIdx.y*BLOCK_DIMENSION_Y*LOOP_Y + threadIdx.y;
	const int col = blockIdx.x*BLOCK_DIMENSION_X*LOOP_X + threadIdx.x;

	const long long thread_id = ((device_id*gridDim.y + blockIdx.y)*gridDim.x + blockIdx.x)*BLOCK_DIMENSION_X*BLOCK_DIMENSION_Y +
	                              threadIdx.y*BLOCK_DIMENSION_X + threadIdx.x;

	hiprandStatePhilox4_32_10_t rng;
	hiprand_init(seed, thread_id, 0, &rng);

	INT2_T __tmp[LOOP_Y][LOOP_X];
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__tmp[i][j] = __mymake_int2(INT_T(0),INT_T(0));
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			#pragma unroll
			for(int k = 0; k < 8*sizeof(INT_T); k += BITXSPIN) {
				#pragma unroll
				for(int l = 0; l < BITXSPIN; l++) {
					if (hiprand_uniform(&rng) < tgtProb) {
						__tmp[i][j].x |= INT_T(1) << (k+l);
					}
					if (hiprand_uniform(&rng) < tgtProb) {
						__tmp[i][j].y |= INT_T(1) << (k+l);
					}
				}
			}
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			hamB[(begY + row+i*BLOCK_DIMENSION_Y)*dimX + col+j*BLOCK_DIMENSION_X] = __tmp[i][j];
		}
	}
	return;
}


template<int BLOCK_DIMENSION_X, int BLOCK_DIMENSION_Y, int LOOP_X, int LOOP_Y, int BITXSPIN, typename INT_T, typename INT2_T>
__global__ void hamiltInitW_k(const int xsl, const int ysl, const long long begY,
		              					  const long long dimX,
		              				    const INT2_T *__restrict__ hamB,
		                    		  INT2_T *__restrict__ hamW)
{
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	const int row = blockIdx.y*BLOCK_DIMENSION_Y*LOOP_Y + tidy;
	const int col = blockIdx.x*BLOCK_DIMENSION_X*LOOP_X + tidx;

	INT2_T __me[LOOP_Y][LOOP_X];

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__me[i][j] = hamB[(begY+row+i*BLOCK_DIMENSION_Y)*dimX + col+j*BLOCK_DIMENSION_X];
		}
	}

	INT2_T __up[LOOP_Y][LOOP_X];
	INT2_T __ct[LOOP_Y][LOOP_X];
	INT2_T __dw[LOOP_Y][LOOP_X];
	INT2_T __sd[LOOP_Y][LOOP_X];

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__up[i][j].x = (__me[i][j].x & 0x8888888888888888ull) >> 1;
			__up[i][j].y = (__me[i][j].y & 0x8888888888888888ull) >> 1;

			__dw[i][j].x = (__me[i][j].x & 0x4444444444444444ull) << 1;
			__dw[i][j].y = (__me[i][j].y & 0x4444444444444444ull) << 1;
		}
	}

	const int readBack = !(row%2); // this kernel reads only BLACK Js

	const int BITXWORD = 8*sizeof(INT_T);

	if (!readBack) {
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {

				__ct[i][j].x = (__me[i][j].x & 0x2222222222222222ull) >> 1;
				__ct[i][j].y = (__me[i][j].y & 0x2222222222222222ull) >> 1;

				__ct[i][j].x |= (__me[i][j].x & 0x1111111111111111ull) << (BITXSPIN+1);
				__ct[i][j].y |= (__me[i][j].x & 0x1111111111111111ull) >> (BITXWORD-BITXSPIN - 1);
				__ct[i][j].y |= (__me[i][j].y & 0x1111111111111111ull) << (BITXSPIN+1);

				__sd[i][j].x = (__me[i][j].y & 0x1111111111111111ull) >> (BITXWORD-BITXSPIN - 1);
				__sd[i][j].y = 0;
			}
		}
	} else {
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {

				__ct[i][j].x = (__me[i][j].x & 0x1111111111111111ull) << 1;
				__ct[i][j].y = (__me[i][j].y & 0x1111111111111111ull) << 1;

				__ct[i][j].y |= (__me[i][j].y & 0x2222222222222222ull) >> (BITXSPIN+1);
				__ct[i][j].x |= (__me[i][j].y & 0x2222222222222222ull) << (BITXWORD-BITXSPIN - 1);
				__ct[i][j].x |= (__me[i][j].x & 0x2222222222222222ull) >> (BITXSPIN+1);

				__sd[i][j].y = (__me[i][j].x & 0x2222222222222222ull) << (BITXWORD-BITXSPIN - 1);
				__sd[i][j].x = 0;
			}
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {

		const int yoff = begY+row + i*BLOCK_DIMENSION_Y;

		const int upOff = ( yoff   %ysl) == 0 ? yoff+ysl-1 : yoff-1;
		const int dwOff = ((yoff+1)%ysl) == 0 ? yoff-ysl+1 : yoff+1;

		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {

			const int xoff = col + j*BLOCK_DIMENSION_X;

			atomicOr(&hamW[yoff*dimX + xoff].x, __ct[i][j].x);
			atomicOr(&hamW[yoff*dimX + xoff].y, __ct[i][j].y);

			atomicOr(&hamW[upOff*dimX + xoff].x, __up[i][j].x);
			atomicOr(&hamW[upOff*dimX + xoff].y, __up[i][j].y);

			atomicOr(&hamW[dwOff*dimX + xoff].x, __dw[i][j].x);
			atomicOr(&hamW[dwOff*dimX + xoff].y, __dw[i][j].y);

			const int sideOff = readBack ? (  (xoff   %xsl) == 0 ? xoff+xsl-1 : xoff-1 ):
						       ( ((xoff+1)%xsl) == 0 ? xoff-xsl+1 : xoff+1);

			atomicOr(&hamW[yoff*dimX + sideOff].x, __sd[i][j].x);
			atomicOr(&hamW[yoff*dimX + sideOff].y, __sd[i][j].y);
		}
	}
	return;
}


template<int BLOCK_DIMENSION_X, int BLOCK_DIMENSION_Y, int TILE_X, int TILE_Y, int FRAME_X, int FRAME_Y, typename INT2_T>
__device__ void loadTile(const int slX, const int slY, const long long begY,
			 									 const long long dimX,
			 								 	 const INT2_T *__restrict__ v,
			       		 				 INT2_T tile[][TILE_X+2*FRAME_X])
{
	const int blkx = blockIdx.x;
	const int blky = blockIdx.y;

	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	const int startX =        blkx*TILE_X;
	const int startY = begY + blky*TILE_Y;

	#pragma unroll
	for(int j = 0; j < TILE_Y; j += BLOCK_DIMENSION_Y) {
		int yoff = startY + j+tidy;

		#pragma unroll
		for(int i = 0; i < TILE_X; i += BLOCK_DIMENSION_X) {
			const int xoff = startX + i+tidx;
			tile[FRAME_Y + j+tidy][FRAME_X + i+tidx] = v[yoff*dimX + xoff];
		}
	}
	if (tidy == 0) {
		int yoff = (startY % slY) == 0 ? startY+slY-1 : startY-1;

		#pragma unroll
		for(int i = 0; i < TILE_X; i += BLOCK_DIMENSION_X) {
			const int xoff = startX + i+tidx;
			tile[0][FRAME_X + i+tidx] = v[yoff*dimX + xoff];
		}

		yoff = ((startY+TILE_Y) % slY) == 0 ? startY+TILE_Y - slY : startY+TILE_Y;

		#pragma unroll
		for(int i = 0; i < TILE_X; i += BLOCK_DIMENSION_X) {
			const int xoff = startX + i+tidx;
			tile[FRAME_Y + TILE_Y][FRAME_X + i+tidx] = v[yoff*dimX + xoff];
		}

		// the other branch in slower so skip it if possible
		if (BLOCK_DIMENSION_X <= TILE_Y) {

			int xoff = (startX % slX) == 0 ? startX+slX-1 : startX-1;

			#pragma unroll
			for(int j = 0; j < TILE_Y; j += BLOCK_DIMENSION_X) {
				yoff = startY + j+tidx;
				tile[FRAME_Y + j+tidx][0] = v[yoff*dimX + xoff];
			}

			xoff = ((startX+TILE_X) % slX) == 0 ? startX+TILE_X - slX : startX+TILE_X;

			#pragma unroll
			for(int j = 0; j < TILE_Y; j += BLOCK_DIMENSION_X) {
				yoff = startY + j+tidx;
				tile[FRAME_Y + j+tidx][FRAME_X + TILE_X] = v[yoff*dimX + xoff];
			}
		} else {
			if (tidx < TILE_Y) {
				int xoff = (startX % slX) == 0 ? startX+slX-1 : startX-1;

				yoff = startY + tidx;
				tile[FRAME_Y + tidx][0] = v[yoff*dimX + xoff];;

				xoff = ((startX+TILE_X) % slX) == 0 ? startX+TILE_X - slX : startX+TILE_X;
				tile[FRAME_Y + tidx][FRAME_X + TILE_X] = v[yoff*dimX + xoff];
			}
		}
	}
	return;
}


template<int BLOCK_DIMENSION_X, int BLOCK_DIMENSION_Y, int LOOP_X, int LOOP_Y, int BITXSPIN, int COLOR, typename INT_T, typename INT2_T>
__global__ void spinUpdateV_2D_k(const int device_id, const long long seed, const int it,
		      const int slX, // sublattice size grid_width of one color (in words)
		      const int slY, // sublattice size grid_height of one color
		      const long long begY,
		      const long long dimX, // ld
		      const float vExp[][5],
		      const INT2_T *__restrict__ jDst,
		      const INT2_T *__restrict__ vSrc,
		            INT2_T *__restrict__ vDst)
{
	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSPIN;

	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	__shared__ INT2_T shTile[BLOCK_DIMENSION_Y*LOOP_Y+2][BLOCK_DIMENSION_X*LOOP_X+2];

	loadTile<BLOCK_DIMENSION_X, BLOCK_DIMENSION_Y,
		 BLOCK_DIMENSION_X * LOOP_X,
		 BLOCK_DIMENSION_Y * LOOP_Y,
		 1, 1, INT2_T>(slX, slY, begY, dimX, vSrc, shTile);

	__shared__ float __shExp[2][5];

	#pragma unroll
	for(int i = 0; i < 2; i += BLOCK_DIMENSION_Y) {
		#pragma unroll
		for(int j = 0; j < 5; j += BLOCK_DIMENSION_X) {
			if (i+tidy < 2 && j+tidx < 5) {
				__shExp[i+tidy][j+tidx] = vExp[i+tidy][j+tidx];
			}
		}
	}
	__syncthreads();

	const int row = blockIdx.y*BLOCK_DIMENSION_Y*LOOP_Y + tidy;
	const int col = blockIdx.x*BLOCK_DIMENSION_X*LOOP_X + tidx;

	const long long thread_id = ((device_id*gridDim.y + blockIdx.y)*gridDim.x + blockIdx.x)*BLOCK_DIMENSION_X*BLOCK_DIMENSION_Y +
	                       threadIdx.y*BLOCK_DIMENSION_X + threadIdx.x;

	INT2_T __me[LOOP_Y][LOOP_X];

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__me[i][j] = vDst[(begY+row+i*BLOCK_DIMENSION_Y)*dimX + col+j*BLOCK_DIMENSION_X];
		}
	}

	INT2_T __up[LOOP_Y][LOOP_X];
	INT2_T __ct[LOOP_Y][LOOP_X];
	INT2_T __dw[LOOP_Y][LOOP_X];

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__up[i][j] = shTile[i*BLOCK_DIMENSION_Y +   tidy][j*BLOCK_DIMENSION_X + 1+tidx];
			__ct[i][j] = shTile[i*BLOCK_DIMENSION_Y + 1+tidy][j*BLOCK_DIMENSION_X + 1+tidx];
			__dw[i][j] = shTile[i*BLOCK_DIMENSION_Y + 2+tidy][j*BLOCK_DIMENSION_X + 1+tidx];
		}
	}

	// BLOCK_DIMENSION_Y is power of two so row parity won't change across loops
	const int readBack = (COLOR == C_BLACK) ? !(row%2) : (row%2);

	INT2_T __sd[LOOP_Y][LOOP_X];

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__sd[i][j] = (readBack) ? shTile[i*BLOCK_DIMENSION_Y + 1+tidy][j*BLOCK_DIMENSION_X +   tidx]:
						  shTile[i*BLOCK_DIMENSION_Y + 1+tidy][j*BLOCK_DIMENSION_X + 2+tidx];
		}
	}

	if (readBack) {
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				__sd[i][j].x = (__ct[i][j].x << BITXSPIN) | (__sd[i][j].y >> (8*sizeof(__sd[i][j].y)-BITXSPIN));
				__sd[i][j].y = (__ct[i][j].y << BITXSPIN) | (__ct[i][j].x >> (8*sizeof(__ct[i][j].x)-BITXSPIN));
			}
		}
	} else {
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				__sd[i][j].y = (__ct[i][j].y >> BITXSPIN) | (__sd[i][j].x << (8*sizeof(__sd[i][j].x)-BITXSPIN));
				__sd[i][j].x = (__ct[i][j].x >> BITXSPIN) | (__ct[i][j].y << (8*sizeof(__ct[i][j].y)-BITXSPIN));
			}
		}
	}

	if (jDst != NULL) {
		INT2_T __J[LOOP_Y][LOOP_X];

		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				__J[i][j] = jDst[(begY+row+i*BLOCK_DIMENSION_Y)*dimX + col+j*BLOCK_DIMENSION_X];
			}
		}

		// apply them
		// the 4 bits of J codify: <upJ, downJ, leftJ, rightJ>
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {

				__up[i][j].x ^= (__J[i][j].x & 0x8888888888888888ull) >> 3;
				__up[i][j].y ^= (__J[i][j].y & 0x8888888888888888ull) >> 3;

				__dw[i][j].x ^= (__J[i][j].x & 0x4444444444444444ull) >> 2;
				__dw[i][j].y ^= (__J[i][j].y & 0x4444444444444444ull) >> 2;

				if (readBack) {
					// __sd[][] holds "left" spins
					// __ct[][] holds "right" spins
					__sd[i][j].x ^= (__J[i][j].x & 0x2222222222222222ull) >> 1;
					__sd[i][j].y ^= (__J[i][j].y & 0x2222222222222222ull) >> 1;

					__ct[i][j].x ^= (__J[i][j].x & 0x1111111111111111ull);
					__ct[i][j].y ^= (__J[i][j].y & 0x1111111111111111ull);
				} else {
					// __ct[][] holds "left" spins
					// __sd[][] holds "right" spins
					__ct[i][j].x ^= (__J[i][j].x & 0x2222222222222222ull) >> 1;
					__ct[i][j].y ^= (__J[i][j].y & 0x2222222222222222ull) >> 1;

					__sd[i][j].x ^= (__J[i][j].x & 0x1111111111111111ull);
					__sd[i][j].y ^= (__J[i][j].y & 0x1111111111111111ull);
				}
			}
		}
	}

	hiprandStatePhilox4_32_10_t rng;
	hiprand_init(seed, thread_id, static_cast<long long>(2*SPIN_X_WORD)*LOOP_X*LOOP_Y*(2*it+COLOR), &rng);

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__ct[i][j].x += __up[i][j].x;
			__dw[i][j].x += __sd[i][j].x;
			__ct[i][j].x += __dw[i][j].x;

			__ct[i][j].y += __up[i][j].y;
			__dw[i][j].y += __sd[i][j].y;
			__ct[i][j].y += __dw[i][j].y;
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			#pragma unroll
			for(int z = 0; z < 8*sizeof(INT_T); z += BITXSPIN) {

				const int2 __src = make_int2((__me[i][j].x >> z) & 0xF,
							     (__me[i][j].y >> z) & 0xF);

				const int2 __sum = make_int2((__ct[i][j].x >> z) & 0xF,
							     (__ct[i][j].y >> z) & 0xF);

				const INT_T ONE = static_cast<INT_T>(1);

				if (hiprand_uniform(&rng) <= __shExp[__src.x][__sum.x]) {
					__me[i][j].x ^= ONE << z;
				}
				if (hiprand_uniform(&rng) <= __shExp[__src.y][__sum.y]) {
					__me[i][j].y ^= ONE << z;
				}
			}
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			vDst[(begY + row+i*BLOCK_DIMENSION_Y)*dimX + col+j*BLOCK_DIMENSION_X] = __me[i][j];
		}
	}
	return;
}


template<int BLOCK_DIMENSION_X, int WSIZE, typename T>
__device__ __forceinline__ T __block_sum(T v)
{
	__shared__ T sh[BLOCK_DIMENSION_X / WSIZE];

	const int lid = threadIdx.x%WSIZE;
	const int wid = threadIdx.x/WSIZE;

	#pragma unroll
	for(int i = WSIZE/2; i; i >>= 1) {
		v += __shfl_down_sync(0xFFFFFFFF, v, i);
	}
	if (lid == 0) sh[wid] = v;

	__syncthreads();
	if (wid == 0) {
		v = (lid < (BLOCK_DIMENSION_X / WSIZE)) ? sh[lid] : 0;

		#pragma unroll
		for(int i = (BLOCK_DIMENSION_X/WSIZE)/2; i; i >>= 1) {
			v += __shfl_down_sync(0xFFFFFFFF, v, i);
		}
	}
	__syncthreads();
	return v;
}

// to be optimized
template<int BLOCK_DIMENSION_X, int BITXSPIN, typename INT_T, typename SUM_T>
__global__ void getMagn_k(const long long n,
			                    const INT_T *__restrict__ v,
			                    SUM_T *__restrict__ sum)
{

	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSPIN;

	const long long nth = static_cast<long long>(blockDim.x)*gridDim.x;
	const long long thread_id = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

	SUM_T __cntP = 0;
	SUM_T __cntN = 0;

	for(long long i = 0; i < n; i += nth) {
		if (i+thread_id < n) {
			const int __c = __mypopc(v[i+thread_id]);
			__cntP += __c;
			__cntN += SPIN_X_WORD - __c;
		}
	}
	__cntP = __block_sum<BLOCK_DIMENSION_X, 32>(__cntP);
	__cntN = __block_sum<BLOCK_DIMENSION_X, 32>(__cntN);

	if (threadIdx.x == 0) {
		atomicAdd(sum+0, __cntP);
		atomicAdd(sum+1, __cntN);
	}
	return;
}


static void countSpins(const int redBlocks,
								       const size_t total_length,
								       const size_t sublattice_length,
								       const unsigned long long *d_black_tiles,
								       const unsigned long long *d_white_tiles,
									     unsigned long long **sum_d,
									     unsigned long long *bsum,
									     unsigned long long *wsum)
{
	CHECK_CUDA(hipMemset(sum_d[0], 0, 2*sizeof(**sum_d)));
	getMagn_k<THREADS, BIT_X_SPIN><<<redBlocks, THREADS>>>(total_length, d_black_tiles, sum_d[0]);
	CHECK_ERROR("getMagn_k");
	CHECK_CUDA(hipDeviceSynchronize());

	bsum[0] = 0;
	wsum[0] = 0;

	unsigned long long sum_h[0][2];

	CHECK_CUDA(hipMemcpy(sum_h[0], sum_d[0], 2*sizeof(**sum_h), hipMemcpyDeviceToHost));
	bsum[0] += sum_h[0][0];
	wsum[0] += sum_h[0][1];

	return;
}


template<int BLOCK_DIMENSION_X,
	 int BITXSPIN,
	 int N_CORR,
   typename INT_T,
	 typename SUM_T>
__global__ void getCorr2D_k(const int corrLen,
												    const long long dimX,
												    const long long dimY,
												    const long long begY,
												    const INT_T *__restrict__ black,
												    const INT_T *__restrict__ white,
													  SUM_T *__restrict__ corr)
{
	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSPIN;

	const int thread_id = threadIdx.x;

	const long long startY = begY + blockIdx.x;

	const int SH_LEN = BLOCK_DIMENSION_X + DIV_UP(N_CORR/2, SPIN_X_WORD);

	__shared__ INT_T __shB[SH_LEN];
	__shared__ INT_T __shW[SH_LEN];
	__shared__ SUM_T __shC[N_CORR];

	#pragma unroll
	for(int j = 0; j < N_CORR; j += BLOCK_DIMENSION_X) {
		if (j+thread_id < N_CORR) {
			__shC[j+thread_id] = 0;
		}
	}

	const int chunkDimX = 2*BLOCK_DIMENSION_X*SPIN_X_WORD;

	for(long long l = 0; l < dimX; l += BLOCK_DIMENSION_X) {

		__syncthreads();
		#pragma unroll
		for(int j = 0; j < SH_LEN; j += BLOCK_DIMENSION_X) {
			if (j+thread_id < SH_LEN) {
				const int off = (l+j+thread_id < dimX) ? l+j+thread_id : l+j+thread_id - dimX;
				__shB[j+thread_id] = black[startY*dimX + off];
				__shW[j+thread_id] = white[startY*dimX + off];
			}
		}
		__syncthreads();

		for(int j = 1; j <= corrLen; j++) {

			SUM_T myCorr = 0;

			for(long long i = thread_id; i < chunkDimX; i += BLOCK_DIMENSION_X) {

				// horiz corr
				const long long myWrdX = (i/2) / SPIN_X_WORD;
				const long long myOffX = (i/2) % SPIN_X_WORD;

				INT_T __tmp = ((startY ^ i) & 1) ? __shW[myWrdX] : __shB[myWrdX];
				const int mySpin = (__tmp >> (myOffX*BITXSPIN)) & 0xF;

				const long long nextX = i+j;

				const long long nextWrdX = (nextX/2) / SPIN_X_WORD;
				const long long nextOffX = (nextX/2) % SPIN_X_WORD;

				__tmp = ((startY ^ nextX) & 1) ? __shW[nextWrdX] : __shB[nextWrdX];
				const int nextSpin = (__tmp >> (nextOffX*BITXSPIN)) & 0xF;

				myCorr += (mySpin == nextSpin) ? SUM_T(1) : SUM_T(-1);

				// vert corr
				const long long nextY = (startY+j >= dimY) ? startY+j-dimY : startY+j;

				__tmp = ((nextY ^ i) & 1) ? white[nextY*dimX + l+myWrdX]:
							    black[nextY*dimX + l+myWrdX];
				const int vertSpin = (__tmp >> (myOffX*BITXSPIN)) & 0xF;

				myCorr += (mySpin == vertSpin) ? SUM_T(1) : SUM_T(-1);
			}

			myCorr = __block_sum<BLOCK_DIMENSION_X, 32>(myCorr);
			if (!thread_id) {
				__shC[j-1] += myCorr;
			}
		}
	}
	__syncthreads();

	#pragma unroll
	for(int j = 0; j < N_CORR; j += BLOCK_DIMENSION_X) {
		if (j+thread_id < N_CORR) {
			atomicAdd(corr + j+thread_id, __shC[j+thread_id]);
		}
	}
	return;
}


template<int BLOCK_DIMENSION_X,
	 int BITXSPIN,
	 int N_CORR,
   typename INT_T,
	 typename SUM_T>
__global__ void getCorr2DRepl_k(const int corrLen,
																const long long dimX,
																const long long begY,
															  const long long slX, // sublattice size grid_width of one color (in words)
															  const long long slY, // sublattice size grid_height of one color
																const INT_T *__restrict__ black,
																const INT_T *__restrict__ white,
																      SUM_T *__restrict__ corr)
{
	const int thread_id = threadIdx.x;
	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSPIN;

	const long long startY = begY + blockIdx.x;
	const long long mySLY = startY / slY;

	const long long NSLX = 2ull*dimX*SPIN_X_WORD / slX;

	const int SH_LEN = BLOCK_DIMENSION_X + DIV_UP(N_CORR/2, SPIN_X_WORD);

	__shared__ INT_T __shB[SH_LEN];
	__shared__ INT_T __shW[SH_LEN];

	__shared__ SUM_T __shC[N_CORR];

	#pragma unroll
	for(int j = 0; j < N_CORR; j += BLOCK_DIMENSION_X) {
		if (j+thread_id < N_CORR) {
			__shC[j+thread_id] = 0;
		}
	}

	const int chunkDimX = MIN(2*BLOCK_DIMENSION_X*SPIN_X_WORD, slX);

	const int slXLD = (slX/2) / SPIN_X_WORD;

	for(long long sl = 0; sl < NSLX; sl++) {

		for(long long l = 0; l < slXLD; l += BLOCK_DIMENSION_X) {

			__syncthreads();
			#pragma unroll
			for(int j = 0; j < SH_LEN; j += BLOCK_DIMENSION_X) {
				if (j+thread_id < SH_LEN) {
					const int off = (l+j+thread_id) % slXLD;
					__shB[j+thread_id] = black[startY*dimX + sl*slXLD + off];
					__shW[j+thread_id] = white[startY*dimX + sl*slXLD + off];
				}
			}
			__syncthreads();

			for(int j = 1; j <= corrLen; j++) {

				SUM_T myCorr = 0;

				for(long long i = thread_id; i < chunkDimX; i += BLOCK_DIMENSION_X) {

					// horiz corr
					const long long myWrdX = (i/2) / SPIN_X_WORD;
					const long long myOffX = (i/2) % SPIN_X_WORD;

					INT_T __tmp = ((startY ^ i) & 1) ? __shW[myWrdX] : __shB[myWrdX];
					const int mySpin = (__tmp >> (myOffX*BITXSPIN)) & 0xF;

					const long long nextX = i+j;

					const long long nextWrdX = (nextX/2) / SPIN_X_WORD;
					const long long nextOffX = (nextX/2) % SPIN_X_WORD;

					__tmp = ((startY ^ nextX) & 1) ? __shW[nextWrdX] : __shB[nextWrdX];
					const int nextSpin = (__tmp >> (nextOffX*BITXSPIN)) & 0xF;

					myCorr += (mySpin == nextSpin) ? SUM_T(1) : SUM_T(-1);

					// vert corr
					const long long nextY = (startY+j >= (mySLY+1)*slY) ? startY+j-slY : startY+j;

					__tmp = ((nextY ^ i) & 1) ? white[nextY*dimX + sl*slXLD + l+myWrdX]:
								    black[nextY*dimX + sl*slXLD + l+myWrdX];
					const int vertSpin = (__tmp >> (myOffX*BITXSPIN)) & 0xF;

					myCorr += (mySpin == vertSpin) ? SUM_T(1) : SUM_T(-1);
				}

				myCorr = __block_sum<BLOCK_DIMENSION_X, 32>(myCorr);
				if (!thread_id) {
					__shC[j-1] += myCorr;
				}
			}
		}
	}
	__syncthreads();

	#pragma unroll
	for(int j = 0; j < N_CORR; j += BLOCK_DIMENSION_X) {
		if (j+thread_id < N_CORR) {
			atomicAdd(corr + j+thread_id, __shC[j+thread_id]);
		}
	}
	return;
}


int main(int argc, char **argv) {

	unsigned long long *d_spins=NULL;
	unsigned long long *d_black_tiles=NULL;
	unsigned long long *d_white_tiles=NULL;

	unsigned long long *hamB_d=NULL;
	unsigned long long *hamW_d=NULL;

	hipEvent_t start, stop;
  float elapsed_time;

	const int SPIN_X_WORD = (8 * sizeof(*d_spins)) / BIT_X_SPIN;

	int grid_width = 2048;
	int grid_height = 2048;

	int total_updates = TOTAL_UPDATES_DEFAULT;

	unsigned long long seed = SEED_DEFAULT;

	float temp  = 0.666f;

	int XSL = 0;
	int YSL = 0;

	if (!grid_width || (grid_width % 2) || ((grid_width / 2) % (SPIN_X_WORD*2*BLOCK_DIMENSION_X_DEFINE*BMULT_X))) {
		fprintf(stderr, "\nPlease specify an grid_width dim multiple of %d\n\n", 2 * SPIN_X_WORD * 2 * BLOCK_DIMENSION_X_DEFINE * BMULT_X);
		exit(EXIT_FAILURE);
	}
	if (!grid_height || (grid_height % (BLOCK_DIMENSION_Y_DEFINE * BMULT_Y))) {
		fprintf(stderr, "\nPlease specify a grid_height dim multiple of %d\n\n", BLOCK_DIMENSION_Y_DEFINE * BMULT_Y);
		exit(EXIT_FAILURE);
	}

	XSL = grid_width;
	YSL = grid_height;

	hipDeviceProp_t props;

	printf("\nUsing GPUs:\n");

	CHECK_CUDA(hipGetDeviceProperties(&props, 0));
	printf("\t%2d (%s, %d SMs, %d th/SM max, CC %d.%d, ECC %s)\n",
		0, props.name, props.multiProcessorCount,
		props.maxThreadsPerMultiProcessor,
		props.major, props.minor,
		props.ECCEnabled?"on":"off");

	printf("\n");

	size_t words_per_row = (grid_width / 2) / SPIN_X_WORD;
	// length of a single color section per GPU
	size_t sublattice_length = static_cast<size_t>(grid_height) * words_per_row;
	// total lattice length
	size_t total_length = 2ull * sublattice_length;

	dim3 grid(DIV_UP(words_per_row / 2, BLOCK_DIMENSION_X_DEFINE * BMULT_X), DIV_UP(grid_height, BLOCK_DIMENSION_Y_DEFINE * BMULT_Y));
	dim3 block(BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE);

	printf("Run configuration:\n");
	printf("\tspin/word: %d\n", SPIN_X_WORD);
	printf("\tspins: %zu\n", total_length * SPIN_X_WORD);
	printf("\tseed: %llu\n", seed);
	printf("\titerations: %d\n", total_updates);
	printf("\tblock (x, y): %d, %d\n", block.x, block.y);
	printf("\ttile  (x, y): %d, %d\n", BLOCK_DIMENSION_X_DEFINE * BMULT_X, BLOCK_DIMENSION_Y_DEFINE * BMULT_Y);
	printf("\tgrid  (x, y): %d, %d\n", grid.x, grid.y);

	printf("\ttemp: %f (%f*T_crit)\n", temp, temp / CRIT_TEMP);

	printf("\n");

	printf("\tlattice size:      %8d x %8d\n", grid_height, grid_width);
	printf("\tlattice shape: 2 x %8d x %8zu (%12zu %s)\n", grid_height, words_per_row, total_length, sizeof(*d_spins) == 4 ? "uints" : "ulls");
	printf("\tmemory: %.2lf MB (%.2lf MB per GPU)\n", (total_length*sizeof(*d_spins))/(1024.0 * 1024.0), sublattice_length * 2 * sizeof(*d_spins) / (1024.0 * 1024.0));

	const int redBlocks = MIN(DIV_UP(total_length, THREADS),
				  (props.maxThreadsPerMultiProcessor/THREADS)*props.multiProcessorCount);

	unsigned long long cntPos;
	unsigned long long cntNeg;
	unsigned long long *sum_d[MAX_GPU];

	CHECK_CUDA(hipMalloc(&d_spins, total_length*sizeof(*d_spins)));
	CHECK_CUDA(hipMemset(d_spins, 0, total_length*sizeof(*d_spins)));

	CHECK_CUDA(hipMalloc(&sum_d[0], 2*sizeof(**sum_d)));


	d_black_tiles = d_spins;
	d_white_tiles = d_spins + total_length/2;

	float *exp_d[MAX_GPU];
	float  exp_h[2][5];

	// precompute possible exponentials
	for(int i = 0; i < 2; i++) {
		for(int j = 0; j < 5; j++) {
			if(temp > 0) {
				exp_h[i][j] = expf((i?-2.0f:2.0f)*static_cast<float>(j*2-4)*(1.0f/temp));
			} else {
				if(j == 2) {
					exp_h[i][j] = 0.5f;
				} else {
					exp_h[i][j] = (i?-2.0f:2.0f)*static_cast<float>(j*2-4);
				}
			}
		}
	}

	CHECK_CUDA(hipSetDevice(0));
	CHECK_CUDA(hipMalloc(exp_d, 2*5*sizeof(**exp_d)));
	CHECK_CUDA(hipMemcpy(exp_d[0], exp_h, 2*5*sizeof(**exp_d), hipMemcpyHostToDevice));


	CHECK_CUDA(hipEventCreate(&start));
	CHECK_CUDA(hipEventCreate(&stop));

	CHECK_CUDA(hipSetDevice(0));
	initialise_traders<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BMULT_X, BMULT_Y, BIT_X_SPIN, C_BLACK, unsigned long long>
	<<<grid, block>>>
	(seed, words_per_row / 2, reinterpret_cast<ulonglong2 *>(d_black_tiles));
	CHECK_ERROR("initialise_traders");

	initialise_traders<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BMULT_X, BMULT_Y, BIT_X_SPIN, C_WHITE, unsigned long long>
	<<<grid, block>>>
	(seed, words_per_row / 2, reinterpret_cast<ulonglong2 *>(d_white_tiles));
	CHECK_ERROR("initialise_traders");

	// computes sum over array
	countSpins(redBlocks, total_length, sublattice_length, d_black_tiles, d_white_tiles, sum_d, &cntPos, &cntNeg);
	printf("\nInitial magnetization: %9.6lf, up_s: %12llu, dw_s: %12llu\n",
	       abs(static_cast<double>(cntPos)-static_cast<double>(cntNeg)) / (total_length*SPIN_X_WORD),
	       cntPos, cntNeg);

	CHECK_CUDA(hipSetDevice(0));
	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipEventRecord(start, 0));
  int iteration;
	// main update loop
	for(iteration = 0; iteration < total_updates; iteration++) {

		CHECK_CUDA(hipSetDevice(0));
		spinUpdateV_2D_k<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BMULT_X, BMULT_Y, BIT_X_SPIN, C_BLACK, unsigned long long>
		<<<grid, block>>>
		(0, seed, iteration + 1, (XSL / 2) / SPIN_X_WORD / 2, YSL, 0, words_per_row / 2,
		 reinterpret_cast<float (*)[5]>(exp_d[0]),
		 reinterpret_cast<ulonglong2 *>(hamW_d),
		 reinterpret_cast<ulonglong2 *>(d_white_tiles),
		 reinterpret_cast<ulonglong2 *>(d_black_tiles));

		CHECK_CUDA(hipSetDevice(0));
		spinUpdateV_2D_k<BLOCK_DIMENSION_X_DEFINE, BLOCK_DIMENSION_Y_DEFINE, BMULT_X, BMULT_Y, BIT_X_SPIN, C_WHITE, unsigned long long>
		<<<grid, block>>>
		(0, seed, iteration + 1, (XSL / 2) / SPIN_X_WORD / 2, YSL, 0, words_per_row / 2,
		 reinterpret_cast<float (*)[5]>(exp_d[0]),
		 reinterpret_cast<ulonglong2 *>(hamB_d),
		 reinterpret_cast<ulonglong2 *>(d_black_tiles),
		 reinterpret_cast<ulonglong2 *>(d_white_tiles));
	}
	CHECK_CUDA(hipEventRecord(stop, 0));
	CHECK_CUDA(hipEventSynchronize(stop));

	// compute total sum
	countSpins(redBlocks, total_length, sublattice_length, d_black_tiles, d_white_tiles, sum_d, &cntPos, &cntNeg);
	printf("Final   magnetization: %9.6lf, up_s: %12llu, dw_s: %12llu (iter: %8d)\n\n",
	       abs(static_cast<double>(cntPos)-static_cast<double>(cntNeg)) / (total_length*SPIN_X_WORD),
	       cntPos, cntNeg, iteration);

	CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

	printf("Kernel execution time for %d update steps: %E ms, %.2lf flips/ns (BW: %.2lf GB/s)\n",
		iteration, elapsed_time, static_cast<double>(total_length*SPIN_X_WORD) * iteration / (elapsed_time * 1.0E+6),
		(2ull * iteration * (
			  sizeof(*d_spins)*((total_length / 2) + (total_length / 2) + (total_length / 2))  // src color read, dst color read, dst color write
			+ sizeof(*exp_d) * 5 * grid.x * grid.y ) * 1.0E-9) / (elapsed_time / 1.0E+3));

	CHECK_CUDA(hipFree(d_spins));


	CHECK_CUDA(hipFree(exp_d[0]));
	CHECK_CUDA(hipFree(sum_d[0]));

  CHECK_CUDA(hipSetDevice(0));
  CHECK_CUDA(hipDeviceReset());

	return 0;
}
